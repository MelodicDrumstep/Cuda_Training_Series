
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t N = 8ULL*1024ULL*1024ULL;  // data size
//const size_t N = 256*640; // data size
const int BLOCK_SIZE = 256;  // CUDA maximum is 1024
// naive atomic reduction kernel
__global__ void atomic_red(const float *gdata, float *out)
{
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < N) 
  {
      atomicAdd(out, gdata[idx]);
      //Just use atomic add to sum all the elements in the array
      //This is not efficient, but it is correct 
      //And nead a lot of threads
  }
}


__global__ void reduce(float * d_A, float * d_sum, size_t N)
{
    __shared__ float sdata[BLOCK_SIZE];
    //Create a shared memory of BLOCK_SIZE
    //Each thread in this block
    //will fill in a single value in this shared memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    sdata[tid] = 0.0f;
    for(int i = idx; i < N; i += blockDim.x * gridDim.x)
    {
        sdata[tid] += d_A[i];
        //This is a grid-stride loop
        //This will add up elements by a stride of blockDim.x * gridDim.x
        //then store it to the shared memory
        //This step reduce the array size from
        //N to blockDim.x * gridDim.x
    }

    //Following is a reduction step:
    //Each time we reduce the array size by half
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();
        //Call __syncthreads first
        //At the first time we can sync from the grid-stride loop
        //And at each loop we can sync from every threads in the block
        //This is to make sure that all threads have finished the previous step
        if(tid < s)
        {
            sdata[tid] += sdata[tid + s];
            //For the first half of the threads
            //We add the second half of the threads
            //This type of memory accesing can be coalesced
        }
    }
    //Af the loop, each block finish its reduction
    //Then for the shared memory of each block
    //the first element store the sum of that block
    if(tid == 0)
    {
        d_sum[blockIdx.x] = sdata[0];
        //Then we output the sum of each block
        //The whole output array is of size "blocks", i.e. "gridDim.x"
    }
}

 __global__ void reduce_a(float *gdata, float *out)
 {
     __shared__ float sdata[BLOCK_SIZE];
     int tid = threadIdx.x;
     sdata[tid] = 0.0f;
     int idx = threadIdx.x + blockDim.x * blockIdx.x;

     while (idx < N) 
     {  // grid stride loop to load data
        sdata[tid] += gdata[idx];
        idx += gridDim.x * blockDim.x;  
      }

     for (unsigned int s=blockDim.x / 2; s > 0; s >>= 1) 
     {
        __syncthreads();
        if (tid < s)  // parallel sweep reduction
        {
            sdata[tid] += sdata[tid + s];
        }
      }
     if (tid == 0) 
     {
      atomicAdd(out, sdata[0]);
     }
  }


__global__ void reduce_ws(float *gdata, float *out)
{
     __shared__ float sdata[32];
     //This 32 is computed by BLOCKSIZE / WARPSIZE
     //namely how many warps are in a single block
     int tid = threadIdx.x;
     int idx = threadIdx.x + blockDim.x*blockIdx.x;
     float val = 0.0f;
     unsigned mask = 0xFFFFFFFFU;
     //This mask means all threads in a warp
     //are participating
     //Then all of them must be sync for warp shuffle
     int lane = threadIdx.x % warpSize;
     //lane means the id of this thread within a warp
     int warpID = threadIdx.x / warpSize;
     //warpID means which warp it's in
     while (idx < N) 
     {  // grid stride loop to load 
        val += gdata[idx];
        idx += gridDim.x * blockDim.x;  
        //Notice that we use a local vairable to 
        //store the sum of the grid-stride loop
        //rather than store it in shared memory
     }

    // 1st warp-shuffle reduction
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) 
    {
       val += __shfl_down_sync(mask, val, offset);
       //shuffle the local variable "val" from 
       //the right "offset" thread to this thread
    }
    //Now the local variable "val" of the id 0 thread in each warp
    //contains the sum of these threads in this warp
    if (lane == 0) 
    {
       sdata[warpID] = val;
    }
    __syncthreads(); //put warp results in shared mem
    //Notice that __syncthread is a block-level sync

    // hereafter, just use all the threads in warp 0 in each block
    //This is because we BLOCKSIZE / WARPSIZE = 32 = WARPSIZE here
    if (warpID == 0)
    {
       // reload val from shared mem if warp existed
       val = (tid < blockDim.x / warpSize) ? sdata[lane] : 0;
       //Here we want to use all the threads within the first warp
       //to continue computing
       //that's because warp shuffle can only be used
       //within a warp

       // final warp-shuffle reduction
       for(int offset = warpSize / 2; offset > 0; offset >>= 1) 
       {
          val += __shfl_down_sync(mask, val, offset);
       }
       if(tid == 0) 
       {
        //every block will issue an atomic add
         atomicAdd(out, val);
       }
   }
}




int main(){

  float *h_A, *h_sum, *d_A, *d_sum;
  h_A = new float[N];  // allocate space for data in host memory
  h_sum = new float;
  for (int i = 0; i < N; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
  hipMalloc(&d_A, N*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sum, sizeof(float));  // allocate device space for sum
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  atomic_red<<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("atomic reduction kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("atomic reduction kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("atomic sum reduction incorrect!\n"); return -1;}
  printf("atomic sum reduction correct!\n");
  const int blocks = 640;
  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  reduce_a<<<blocks, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("reduction w/atomic kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("reduction w/atomic kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("reduction w/atomic sum incorrect!\n"); return -1;}
  printf("reduction w/atomic sum correct!\n");
  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  reduce_ws<<<blocks, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("reduction warp shuffle kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("reduction warp shuffle kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("reduction warp shuffle sum incorrect!\n"); return -1;}
  printf("reduction warp shuffle sum correct!\n");
  return 0;
}
  
